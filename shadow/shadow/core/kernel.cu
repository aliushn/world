#include "hip/hip_runtime.h"
#include "kernel.hpp"
#include "util/log.hpp"

#include <cmath>

namespace Shadow {

namespace Kernel {

#if defined(USE_CUDA)
void Synchronize() { CUDA_CHECK(hipDeviceSynchronize()); }

template <typename T, typename Dtype>
T *MakeBuffer(size_t size, Dtype *host_ptr) {
  T *buffer;
  CUDA_CHECK(hipMalloc(&buffer, size * sizeof(Dtype)));
  if (host_ptr != nullptr) {
    WriteBuffer(size, host_ptr, buffer);
  }
  return buffer;
}

template <typename T, typename Dtype>
void ReadBuffer(size_t size, const T *src, Dtype *des) {
  CUDA_CHECK(
      hipMemcpy(des, src, size * sizeof(Dtype), hipMemcpyDeviceToHost));
}

template <typename T, typename Dtype>
void WriteBuffer(size_t size, const Dtype *src, T *des) {
  CUDA_CHECK(
      hipMemcpy(des, src, size * sizeof(Dtype), hipMemcpyHostToDevice));
}

template <typename T, typename Dtype>
void CopyBuffer(size_t size, const T *src, T *des) {
  CUDA_CHECK(
      hipMemcpy(des, src, size * sizeof(Dtype), hipMemcpyDeviceToDevice));
}

template <typename T>
void ReleaseBuffer(T *buffer) {
  CUDA_CHECK(hipFree(buffer));
}

// Explicit instantiation
template int *MakeBuffer<int, int>(size_t size, int *host_ptr);
template float *MakeBuffer<float, float>(size_t size, float *host_ptr);
template unsigned char *MakeBuffer<unsigned char, unsigned char>(
    size_t size, unsigned char *host_ptr);

template void ReadBuffer<int, int>(size_t size, const int *src, int *des);
template void ReadBuffer<float, float>(size_t size, const float *src,
                                       float *des);
template void ReadBuffer<unsigned char, unsigned char>(size_t size,
                                                       const unsigned char *src,
                                                       unsigned char *des);

template void WriteBuffer<int, int>(size_t size, const int *src, int *des);
template void WriteBuffer<float, float>(size_t size, const float *src,
                                        float *des);
template void WriteBuffer<unsigned char, unsigned char>(
    size_t size, const unsigned char *src, unsigned char *des);

template void CopyBuffer<int, int>(size_t size, const int *src, int *des);
template void CopyBuffer<float, float>(size_t size, const float *src,
                                       float *des);
template void CopyBuffer<unsigned char, unsigned char>(size_t size,
                                                       const unsigned char *src,
                                                       unsigned char *des);

template void ReleaseBuffer<int>(int *buffer);
template void ReleaseBuffer<float>(float *buffer);
template void ReleaseBuffer<unsigned char>(unsigned char *buffer);
#endif

}  // namespace Kernel

}  // namespace Shadow
